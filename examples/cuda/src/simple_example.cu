
#include <hip/hip_runtime.h>
#include <iostream>

namespace shunya::cuda::examples {

// Kernel definition
__global__ void VecAdd(float* vector_a, float* vector_b, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    vector_b[i] = vector_a[i] + vector_b[i];
  }
}

void RunVecAddExample() {
  int N = 1<<20; // 1M elements

  float *x = new float[N];
  float *y = new float[N];

  // Allocate Unified Memory – accessible from CPU or GPU
  hipError_t allocErr;
  allocErr = hipMallocManaged(&x, N * sizeof(float));
  if (allocErr != hipSuccess) {
    std::cerr << "Failed to allocate memory for x: " << hipGetErrorString(allocErr) << std::endl;
    return;
  }
  allocErr = hipMallocManaged(&y, N * sizeof(float));
  if (allocErr != hipSuccess) {
    std::cerr << "Failed to allocate memory for y: " << hipGetErrorString(allocErr) << std::endl;
    return;
  }

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  VecAdd<<<1, 1>>>(x, y, N);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    return;
  }

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
}

} // namespace shunya::cuda::examples